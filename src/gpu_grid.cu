#include "hip/hip_runtime.h"
#ifndef GPU_GRID_H_
#define GPU_GRID_H_

#include "gpu_grid.cuh"
#include "gpu_kernels.cuh"
#include "hip/hip_runtime_api.h"
#include <stdio.h>

extern "C"
{

namespace bdm {

// void copyArrayToDevice(void *device, const void *host, int offset, int size)
// {
//   checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
// }

// void ParticleSystem::setArray(const float *data, int start, int count) {
//   copyArrayToDevice(m_dVel, data, start*4*sizeof(float), count*4*sizeof(float));
// }

//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b) {
  return (a % b != 0) ? (a / b + 1) : (a / b);
}

void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads) {
  numThreads = min(blockSize, n);
  numBlocks = iDivUp(n, numThreads);
}

void calculate_hash(uint  *gridParticleHash,
                  uint  *gridParticleIndex,
                  float *pos,
                  int    numParticles) {
  uint numThreads, numBlocks;
  computeGridSize(numParticles, 64, numBlocks, numThreads);

  // hello_world<<<5,5>>>();

  printf("Launching calculate_hash_d<<<%d, %d>>>...\n", numBlocks, numThreads);

  // execute the kernel
  calculate_hash_d<<< numBlocks, numThreads >>>(gridParticleHash,
                                         gridParticleIndex,
                                         (float4 *) pos,
                                         numParticles);

  // check if kernel invocation generated an error
  getLastCudaError("Kernel execution failed");
  hipDeviceReset();
}
}  // namespace bdm
}

#endif  // GPU_GRID_H_
